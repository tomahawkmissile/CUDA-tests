
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>

//This program adds a vector array of FP32 and FP64 types on a single GPU thread.

__global__ void run32_gpu(float* out, float* a, float* b, int n) {
    for(unsigned int i=0;i<n;i++) {
        out[i]=a[i]+b[i];
    }
}
__global__ void run64_gpu(double* out, double* a, double* b, int n) {
    for(unsigned int i=0;i<n;i++) {
        out[i]=a[i]+b[i];
    }
}
__global__ void run32_parallel(float* out, float* a, float* b, int n) {
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    if(n<xid) {
        out[xid]=a[xid]+b[xid];
    }
}
__global__ void run64_parallel(double* out, double* a, double* b, int n) {
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    if(n<xid) {
        out[xid]=a[xid]+b[xid];
    }
}

int main(int argc, char* argv[]) {

    if(argc!=2) {
        printf("Only pass 1 argument, which is the amount of array elements.\n");
        return -1;
    }

    int n=0;

    sscanf(argv[1],"%i",&n);
    printf("Starting with %i array elements.\n", n);

    float *fa, *fb, *fout, *gfa, *gfb, *gfout;
    double *da, *db, *dout, *gda, *gdb, *gdout; 

    fa=(float*)malloc(sizeof(float)*n);
    fb=(float*)malloc(sizeof(float)*n);
    fout=(float*)malloc(sizeof(float)*n);
    
    da=(double*)malloc(sizeof(double)*n);
    db=(double*)malloc(sizeof(double)*n);
    dout=(double*)malloc(sizeof(double)*n);

    for(unsigned int i=0;i<n;i++) {
        fa[i]=1.0;
        fb[i]=(float)i;
        
        da[i]=1.0;
        db[i]=(double)i;
    }

    hipMalloc((void**)&gfa, sizeof(float)*n);
    hipMalloc((void**)&gfb, sizeof(float)*n);
    hipMalloc((void**)&gda, sizeof(double)*n);
    hipMalloc((void**)&gdb, sizeof(double)*n);

    hipMalloc((void**)&gfout, sizeof(float)*n);
    hipMalloc((void**)&gdout, sizeof(double)*n);

    hipMemcpy(gfa, fa, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(gfb, fb, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(gda, da, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(gdb, db, sizeof(double)*n, hipMemcpyHostToDevice);

    clock_t f_start = clock();
    run32_gpu<<<1,1>>>(gfout, gfa, gfb, n);
    clock_t f_end = clock();
    printf("FP32 single-thread test took: %f\n",((double)(f_end-f_start))/CLOCKS_PER_SEC);

    clock_t d_start = clock();
    run64_gpu<<<1,1>>>(gdout, gda, gdb, n);
    clock_t d_end = clock();
    printf("FP64 single-thread test took: %f\n",((double)(d_end-d_start))/CLOCKS_PER_SEC);


    f_start = clock();
    run32_parallel<<<1,256>>>(gfout, gfa, gfb, n);
    f_end = clock();
    printf("FP32 single-thread test took: %f\n",((double)(f_end-f_start))/CLOCKS_PER_SEC);

    d_start = clock();
    run64_parallel<<<1,256>>>(gdout, gda, gdb, n);
    d_end = clock();
    printf("FP64 single-thread test took: %f\n",((double)(d_end-d_start))/CLOCKS_PER_SEC);


    printf("Done.\n");

    hipFree(gfa);
    hipFree(gfb);
    hipFree(gfout);
    hipFree(gda);
    hipFree(gdb);
    hipFree(gdout);

    free(fa);
    free(fb);
    free(fout);
    free(da);
    free(db);
    free(dout);

    return 0;
}
