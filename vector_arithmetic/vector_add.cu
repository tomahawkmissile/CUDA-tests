#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>

//This program adds a vector array of FP32 and FP64 types on a single CPU core.

__global__ void run32_gpu(float* out, float* a, float* b, int n) {
    for(unsigned int i=0;i<n;i++) {
        out[i]=a[i]+b[i];
    }
}
__global__ void run64_gpu(float* out, float* a, float* b, int n) {
    for(unsigned int i=0;i<n;i++) {
        out[i]=a[i]+b[i];
    }
}

int main(int argc, char* argv[]) {

    if(argc!=1) {
        printf("Only pass 1 argument, which is the amount of array elements.\n");
        return -1;
    }
    sscanf(argv[0],"%i",&n);
    printf("Starting with %i array elements.\n", n);

    int n;

    float *fa, *fb, *fout, *gfa, *gfb, *gfout;
    double *da, *db, *dout, *gda, *gdb, *gdout; 

    fa=(float*)malloc(sizeof(float)*n);
    fb=(float*)malloc(sizeof(float)*n);
    fout=(float*)malloc(sizeof(float)*n);
    
    da=(double*)malloc(sizeof(double)*n);
    db=(double*)malloc(sizeof(double)*n);
    dout=(double*)malloc(sizeof(double)*n);

    for(unsigned int i=0;i<n;i++) {
        fa[i]=1.0;
        fb[i]=(float)i;
        
        da[i]=1.0;
        db[i]=(double)i;
    }

    hipMalloc((void**)&gfa, sizeof(float)*n);
    hipMalloc((void**)&gfb, sizeof(float)*n);
    hipMalloc((void**)&gda, sizeof(double)*n);
    hipMalloc((void**)&gdb, sizeof(double)*n);

    hipMemcpy(gfa, fa, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(gfb, fb, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(gda, da, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(gdb, db, sizeof(double)*n, hipMemcpyHostToDevice);

    clock_t f_start = clock();
    run32_gpu<<<1,1>>>(gfout, gfa, gfb, n);
    clock_t f_end = clock();
    printf("FP32 test took: %f\n",((double)(f_end-f_start))/CLOCKS_PER_SEC);

    clock_t d_start = clock();
    run64_gpu<<<1,1>>>(gdout, gda, gdb, n);
    clock_t d_end = clock();
    printf("FP64 test took: %f\n",((double)(d_end-d_start))/CLOCKS_PER_SEC);

    printf("Done.\n");

    return 0;
}